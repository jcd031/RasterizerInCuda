#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <cmath>

#include "../include/obj.hpp"

// Project takes some 3D coordinates and transform them
// in 2D coordinates using the transformation matrix
__host__ __device__
void Project(Vec3f coord, Vec3i &ret, Matrix44f transMat, const uint32_t imageWidth, const uint32_t imageHeight, const uint32_t widthOffset, const uint32_t heightOffset)
{
   ret.z = coord.x * transMat.x[0][2] + coord.y * transMat.x[1][2] + coord.z * transMat.x[2][2] + transMat.x[3][2];
   float pCamerax = coord.x * transMat.x[0][0] + coord.y * transMat.x[1][0] + coord.z * transMat.x[2][0] + transMat.x[3][0];
   float pCameray = coord.x * transMat.x[0][1] + coord.y * transMat.x[1][1] + coord.z * transMat.x[2][1] + transMat.x[3][1];

   ret.x = (int)((pCamerax / -ret.z) * imageHeight + widthOffset); 
   ret.y = (int)((pCameray / ret.z) * imageHeight + heightOffset);
}

__host__ __device__
void setPixel(
      uint32_t * pixels,
      const int x,
      const int y,
      const int color,
      const uint32_t imageWidth,
      const uint32_t imageHeight
      )
{
   if (x >=0 && x < imageWidth && y >=0 && y < imageHeight)
   {
      int index = y * imageWidth + x;
      pixels[index] = color;
   }
}

// Clamping values to keep them between 0 and 1
__host__ __device__
float Clamp(float value, float min = 0, float max = 1)
{
   return (min > ((value < max) ? value:max)) ? min:value;
   //return Math.Max(min, Math.Min(value, max));
}

// Interpolating the value between 2 vertices 
// min is the starting point, max the ending point
// and gradient the % between the 2 points
__host__ __device__
float Interpolate(float min, float max, float gradient)
{
   return min + (max - min) * Clamp(gradient);
}

struct triangleData
{
   int dXa;
   int dYa;
   int dYb;
   int dXb;
   int dXc;
   int dYc;

   int top;
   int left;
   int bottom;
   int right;

   int eA;
   int eB;
   int eC;
   
   int boxHeight;
   int boxWidth;
   int boxSize;
};

__global__
void computeTriangles( 
      uint32_t * pixels,
      triangleData * tD,
      const Vec3f *verts,
      const int * tris,
      const int numTris,
      Vec3i * projected,
      const Matrix44f worldToCamera,
      const uint32_t imageWidth,
      const uint32_t imageHeight
      )
{
   int indexGPU = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;
   for (int i = indexGPU; i < numTris; i+=stride)
   {
      // Get triangle data
      Vec3i A = projected[tris[i]], B = projected[tris[i+1]], C = projected[tris[i+2]];

      tD[i].top = max(0, min(A.y, min(B.y, C.y)));
      tD[i].left = max(0, min(A.x, min(B.x, C.x)));
      tD[i].bottom = min(imageHeight, max(A.y, max(B.y, C.y)));
      tD[i].right = min(imageWidth, max(A.x, max(B.x, C.x)));

      tD[i].boxHeight = tD[i].bottom - tD[i].top;
      tD[i].boxWidth = tD[i].right - tD[i].left;
      if(tD[i].boxHeight > 0 && tD[i].boxWidth > 0)
      {
         tD[i].boxSize = tD[i].boxHeight * tD[i].boxWidth;

         tD[i].dXa = B.x - A.x;
         tD[i].dYa = B.y - A.y;
         tD[i].dYb = C.y - B.y;
         tD[i].dXb = C.x - B.x;
         tD[i].dXc = A.x - C.x;
         tD[i].dYc = A.y - C.y;

         tD[i].eA = (tD[i].left - A.x) * tD[i].dYa - (tD[i].top - A.y) * tD[i].dXa;
         tD[i].eB = (tD[i].left - B.x) * tD[i].dYb - (tD[i].top - B.y) * tD[i].dXb;
         tD[i].eC = (tD[i].left - C.x) * tD[i].dYc - (tD[i].top - C.y) * tD[i].dXc;
      }
      else
      {
         tD[i].boxSize = 0;
      }
   }
}

__global__
void rasterTriangles(
      uint32_t * pixels,
      const triangleData * tD,
      const int numTris,
      const uint32_t imageWidth,
      const uint32_t imageHeight
      )
{
   // Temporary debugging color
   //int color = 0x00ffff + 255 * blockIdx.x / blockDim.x;
   int color = 0xffffff;

   for ( int i = 0; i < numTris; i++)
   {
      if ( tD[i].boxSize > 0)
      {
         int indexGPU = blockIdx.x * blockDim.x + threadIdx.x;
         int stride = blockDim.x * gridDim.x;
         for (int j = indexGPU; j < tD[i].boxSize; j+=stride)
         {
            int x = j % tD[i].boxWidth;
            int y = j / tD[i].boxWidth;

            int offsetA = x * tD[i].dYa - y * tD[i].dXa;
            int offsetB = x * tD[i].dYb - y * tD[i].dXb;
            int offsetC = x * tD[i].dYc - y * tD[i].dXc;

            if (tD[i].eA + offsetA >= 0 && tD[i].eB + offsetB >= 0 & tD[i].eC + offsetC >= 0)
            {
               setPixel(pixels, tD[i].left + x, tD[i].top + y, color, imageWidth, imageHeight);
            }
         }
      }
   }
}
// Compute the 2D pixel coordinates of a point defined in world space. This function
// requires the point original world coordinates of course, the world-to-camera
// matrix (which you can get from computing the inverse of the camera-to-world matrix,
// the matrix transforming the camera), the canvas dimension and the image width and
// height in pixels.
__global__
void computePixelCoordinates( 
      uint32_t * pixels,
      const Vec3f *verts,
      const int n,
      Vec3i *pRaster,
      const Matrix44f worldToCamera,
      const uint32_t imageHeight,
      const uint32_t imageWidth,
      const int wXh,
      const uint32_t widthOffset,
      const uint32_t heightOffset
      )
{
   int indexGPU = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;
   for (int i = indexGPU; i < n; i+=stride)
   {
      float pCameraz = verts[i].x * worldToCamera.x[0][2] + verts[i].y * worldToCamera.x[1][2] + verts[i].z * worldToCamera.x[2][2] + worldToCamera.x[3][2];
      if(pCameraz < -1)
      {
         float pCamerax = verts[i].x * worldToCamera.x[0][0] + verts[i].y * worldToCamera.x[1][0] + verts[i].z * worldToCamera.x[2][0] + worldToCamera.x[3][0];
         float pCameray = verts[i].x * worldToCamera.x[0][1] + verts[i].y * worldToCamera.x[1][1] + verts[i].z * worldToCamera.x[2][1] + worldToCamera.x[3][1];

         pRaster[i].x = (int)((pCamerax / -pCameraz) * imageHeight + widthOffset); 
         pRaster[i].y = (int)((pCameray / pCameraz) * imageHeight + heightOffset);

         //setPixel(pixels, pRaster[i].x, pRaster[i].y, 0xffffff, imageWidth, imageHeight);
      }
   }
}

Vec3f *verts;
Vec3i *vertArray;
int *tris;
triangleData * tD;

int numTris;
int numVertices;

obj * xtree;

uint32_t imageWidth;
uint32_t imageHeight;
uint32_t * pixels;

   extern "C"
int init(const uint32_t iW, const uint32_t iH, uint32_t ** p)
{
   imageWidth = iW;
   imageHeight = iH;

   // Magically knowing the size of each obj beforehand. Will need to create a table to grab these values from.
   numTris = 384;
   numVertices = 146;

   hipMallocManaged(&vertArray, numVertices*sizeof(Vec3i));
   hipMallocManaged(&tD, numTris*sizeof(triangleData));
   hipMallocManaged(&tris, numTris*sizeof(int));
   hipMallocManaged(&verts, numVertices*sizeof(Vec3f));
   hipMallocManaged(p, imageWidth*imageHeight*sizeof(uint32_t));
   pixels = *p;

   // Have to set the data after the malloc because hipMalloc resets the pointer;
   *xtree = obj("xtree.obj", verts, tris);

   return 0;
}

   extern "C"
int run(Matrix44f worldToCamera)
{
   int wXh = imageWidth*imageHeight;
   int widthHalf = imageWidth/2;
   int heightHalf = imageHeight/2;

   int blockSize = 256;
   int numBlocks = (numTris + blockSize - 1) / blockSize;
   computePixelCoordinates<<<numBlocks, blockSize>>>(pixels, verts, numVertices, vertArray, worldToCamera, imageHeight, imageWidth, wXh, widthHalf, heightHalf);

   computeTriangles<<<numBlocks, blockSize>>>(pixels, tD, verts, tris, numTris, vertArray, worldToCamera, imageWidth, imageHeight);
   
   rasterTriangles<<<numTris, blockSize>>>(pixels, tD, numTris, imageWidth, imageHeight);

   hipDeviceSynchronize();

   return 0;
}

   extern "C"
int clearScreen()
{
   hipMemset(pixels, 0, imageWidth*imageHeight*sizeof(uint32_t));
   return 0;
}

   extern "C"
int finish()
{
   hipFree(pixels);
   hipFree(vertArray);
   hipFree(tD);
   hipFree(tris);
   hipFree(verts);

   return 0;
}
